#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include <algorithm>
#include <cassert>
#include <fstream>
#include <functional>
#include <iostream>
#include <iterator>
#include <list>
#include <memory>
#include <mutex>
#include <vector>
#include "neural/factory.h"
#include "neural/writer.h"
#include "utils/bititer.h"
#include "utils/exception.h"
#include "utils/filesystem.h"

#include <algorithm>
#include <cstdio>
#include "NvInfer.h"
#include "neural/network_cuda.h"

namespace lczero {

class Logger : public nvinfer1::ILogger {
 public:
  Logger() : Logger(Severity::kWARNING) {}

  Logger(Severity severity) : reportableSeverity(severity) {}

  void log(Severity severity, const char* msg) override {
    // suppress messages with severity enum value greater than the reportable
    if (severity > reportableSeverity) {
      // lot of messages, no need to spam end-user
      // printf("\nTRT message: %s\n", msg);
    } else {
      char message[128];
      sprintf(message, "TRT error, severity: %d, message: %s", int(severity),
              msg);
      throw Exception(message);
    }
  }

  Severity reportableSeverity{Severity::kINFO};
};

class Int8EntropyCalibrator : public nvinfer1::IInt8EntropyCalibrator {
 public:
  Int8EntropyCalibrator(const std::string& dir) : dir_(dir) {
    file_list_ = GetFileList(dir);
    record_size_ = kInputPlanes * 8 * 8 * sizeof(float);
    batch_.resize(record_size_ * getBatchSize());
    ReportCUDAErrors(hipMalloc(&device_input_, record_size_ * getBatchSize()));
  }

  virtual ~Int8EntropyCalibrator() { ReportCUDAErrors(hipFree(device_input_)); }

  int getBatchSize() const override { return 1024; }

  void toPlanes(const V3TrainingData& v3, float* data) {
    int i;
    for (i = 0; i < 104; i++) {
      for (int j = 63; j >= 0; j--) {
        data[i * 64 + (63 - j)] =
            static_cast<float>((v3.planes[i] >> j) & 1ull);
      }
    }

    for (int j = 0; j < 64; j++)
      data[i * 64 + j] = static_cast<float>(v3.castling_us_ooo);
    i++;
    for (int j = 0; j < 64; j++)
      data[i * 64 + j] = static_cast<float>(v3.castling_us_oo);
    i++;
    for (int j = 0; j < 64; j++)
      data[i * 64 + j] = static_cast<float>(v3.castling_them_ooo);
    i++;
    for (int j = 0; j < 64; j++)
      data[i * 64 + j] = static_cast<float>(v3.castling_them_ooo);
    i++;
    for (int j = 0; j < 64; j++)
      data[i * 64 + j] = static_cast<float>(v3.side_to_move);
    i++;
    for (int j = 0; j < 64; j++) data[i * 64 + j] = 0.0f;
    i++;
    for (int j = 0; j < 64; j++) data[i * 64 + j] = v3.rule50_count / 99.0f;
    i++;
    for (int j = 0; j < 64; j++) data[i * 64 + j] = 1.0f;
    i++;

    assert(i == kInputPlanes);
  }

  bool getBatch(void* bindings[], const char* names[],
                int nbBindings) override {
    std::cerr << "batch " << calls_ + 1 << "/" << total_calls_ << std::endl;
    gzFile infile;
    int rec_num = 0;
    bool sampling = true;

    while (sampling) {
      std::cerr << file_list_[file_num_] << " " << rec_num + 1 << "/"
                << getBatchSize() << std::endl;
      infile = gzopen((dir_ + "/" + file_list_[file_num_]).c_str(), "rb");
      while (gzread(infile, reinterpret_cast<char*>(&v3_), sizeof(v3_)) > 0) {
        toPlanes(v3_, reinterpret_cast<float*>(batch_.data() +
                                               rec_num * record_size_));
        rec_num++;
        if (rec_num == getBatchSize()) {
          sampling = false;
          break;
        }
      }
      gzclose(infile);
      file_num_++;
      file_num_ %= file_list_.size();
    }

    ReportCUDAErrors(hipMemcpy(device_input_, batch_.data(),
                                record_size_ * getBatchSize(),
                                hipMemcpyHostToDevice));
    bindings[0] = device_input_;
    return ++calls_ < total_calls_;
  }

  const void* readCalibrationCache(size_t& length) override {
    calibration_cache_.clear();
    std::ifstream input(calibrationTableName(), std::ios::binary);
    input >> std::noskipws;
    if (read_cache_ && input.good())
      std::copy(std::istream_iterator<char>(input),
                std::istream_iterator<char>(),
                std::back_inserter(calibration_cache_));

    length = calibration_cache_.size();
    return length ? &calibration_cache_[0] : nullptr;
  }

  void writeCalibrationCache(const void* cache, size_t length) override {
    std::ofstream output(calibrationTableName(), std::ios::binary);
    output.write(reinterpret_cast<const char*>(cache), length);
  }

 private:
  static std::string calibrationTableName() {
    return std::string("INT8-Calibration");
  }

  bool read_cache_{true};
  size_t record_size_;
  int calls_{0};
  int total_calls_{100};
  int file_num_{0};
  void* device_input_{nullptr};
  std::vector<char> batch_;
  std::vector<char> calibration_cache_;
  std::vector<std::string> file_list_;
  std::string dir_;
  V3TrainingData v3_;
};

class TRTNetwork;

class TRTNetworkComputation : public NetworkComputation {
 public:
  TRTNetworkComputation(TRTNetwork* network);
  ~TRTNetworkComputation();

  void AddInput(InputPlanes&& input) override {
    auto iter_mask =
        &inputs_outputs_->input_masks_mem_[batch_size_ * kInputPlanes];
    auto iter_val =
        &inputs_outputs_->input_val_mem_[batch_size_ * kInputPlanes];

    int i = 0;
    for (const auto& plane : input) {
      iter_mask[i] = plane.mask;
      iter_val[i] = plane.value;
      i++;
    }

    batch_size_++;
  }

  void ComputeBlocking() override;

  int GetBatchSize() const override { return batch_size_; }

  float GetQVal(int sample) const override {
    return inputs_outputs_->op_value_mem_[sample];
  }

  float GetPVal(int sample, int move_id) const override {
    return inputs_outputs_->op_policy_mem_[sample * kNumOutputPolicy + move_id];
  }

 private:
  // Memory holding inputs, outputs.
  std::unique_ptr<InputsOutputs> inputs_outputs_;
  int batch_size_;

  TRTNetwork* network_;
};  // TRTNetworkComputation

class TRTNetwork : public Network {
 public:
  TRTNetwork(Weights weights, const OptionsDict& options) {
    Int8EntropyCalibrator calibrator(
        "/home/fhuizing/Downloads/chess/test-10/training-20180813-1817");
    gpu_id_ = options.GetOrDefault<int>("gpu", 0);

    int total_gpus;
    ReportCUDAErrors(hipGetDeviceCount(&total_gpus));

    if (gpu_id_ >= total_gpus)
      throw Exception("Invalid GPU Id: " + std::to_string(gpu_id_));

    // Select GPU to run on (for *the current* thread).
    ReportCUDAErrors(hipSetDevice(gpu_id_));

    const int kNumFilters = weights.input.biases.size();
    numBlocks_ = weights.residual.size();

    // 0. Process weights.
    processConvBlock(weights.input, true);
    for (auto i = size_t{0}; i < numBlocks_; i++) {
      processConvBlock(weights.residual[i].conv1, true);
      processConvBlock(weights.residual[i].conv2, true);
    }
    processConvBlock(weights.policy, true);
    processConvBlock(weights.value, true);

    // 1. allocate scratch space used to expand the input planes
    ReportCUDAErrors(hipMalloc(&scratch_mem_, 128 * 1024 * 1024));

    // 2. Build the network
    static Logger gLogger;
    nvinfer1::IBuilder* builder = nvinfer1::createInferBuilder(gLogger);
    nvinfer1::INetworkDefinition* network = builder->createNetwork();

    // Create input tensor of shape { 112, 8, 8 } with name "board"
    nvinfer1::ITensor* inputLayer =
        network->addInput("board", nvinfer1::DataType::kFLOAT,
                          nvinfer1::Dims3{kInputPlanes, 8, 8});
    assert(inputLayer);

    // input conv
    const nvinfer1::Weights inputConvWeights{
        nvinfer1::DataType::kFLOAT, &weights.input.weights[0],
        kNumFilters * kInputPlanes * 3 * 3};
    const nvinfer1::Weights inputConvBiases{
        nvinfer1::DataType::kFLOAT, &weights.input.biases[0], kNumFilters};
    auto inputConv = network->addConvolution(*inputLayer, kNumFilters,
                                             nvinfer1::DimsHW{3, 3},
                                             inputConvWeights, inputConvBiases);
    inputConv->setPadding(nvinfer1::DimsHW{1, 1});
    auto inputConvRelu = network->addActivation(
        *inputConv->getOutput(0), nvinfer1::ActivationType::kRELU);

    nvinfer1::ITensor* lastLayer = inputConvRelu->getOutput(0);

    // residual block
    for (int block = 0; block < numBlocks_; block++) {
      const nvinfer1::Weights conv1Weights{
          nvinfer1::DataType::kFLOAT, &weights.residual[block].conv1.weights[0],
          kNumFilters * kNumFilters * 3 * 3};
      const nvinfer1::Weights conv1Biases{
          nvinfer1::DataType::kFLOAT, &weights.residual[block].conv1.biases[0],
          kNumFilters};
      auto conv1 = network->addConvolution(*lastLayer, kNumFilters,
                                           nvinfer1::DimsHW{3, 3}, conv1Weights,
                                           conv1Biases);
      conv1->setPadding(nvinfer1::DimsHW{1, 1});
      auto relu1 = network->addActivation(*conv1->getOutput(0),
                                          nvinfer1::ActivationType::kRELU);

      const nvinfer1::Weights conv2Weights{
          nvinfer1::DataType::kFLOAT, &weights.residual[block].conv2.weights[0],
          kNumFilters * kNumFilters * 3 * 3};
      const nvinfer1::Weights conv2Biases{
          nvinfer1::DataType::kFLOAT, &weights.residual[block].conv2.biases[0],
          kNumFilters};
      auto conv2 = network->addConvolution(*relu1->getOutput(0), kNumFilters,
                                           nvinfer1::DimsHW{3, 3}, conv2Weights,
                                           conv2Biases);
      conv2->setPadding(nvinfer1::DimsHW{1, 1});

      auto residualAdd =
          network->addElementWise(*conv2->getOutput(0), *lastLayer,
                                  nvinfer1::ElementWiseOperation::kSUM);
      auto relu2 = network->addActivation(*residualAdd->getOutput(0),
                                          nvinfer1::ActivationType::kRELU);

      lastLayer = relu2->getOutput(0);
    }

    nvinfer1::ITensor* resiOutTensor = lastLayer;
    resiOutTensor->setName("resiOut");

    // policy head
    const int kpolicyConvFilters = weights.policy.bn_means.size();
    const nvinfer1::Weights convPolWeights{
        nvinfer1::DataType::kFLOAT, &weights.policy.weights[0],
        kpolicyConvFilters * kNumFilters * 1 * 1};
    const nvinfer1::Weights convPolBiases{nvinfer1::DataType::kFLOAT,
                                          &weights.policy.biases[0],
                                          kpolicyConvFilters};
    auto convPol = network->addConvolution(*resiOutTensor, kpolicyConvFilters,
                                           nvinfer1::DimsHW{1, 1},
                                           convPolWeights, convPolBiases);
    auto reluPol = network->addActivation(*convPol->getOutput(0),
                                          nvinfer1::ActivationType::kRELU);

    const int knumPolicyOutputs = weights.ip_pol_b.size();
    const nvinfer1::Weights fcPolWeights{
        nvinfer1::DataType::kFLOAT, &weights.ip_pol_w[0],
        knumPolicyOutputs * kpolicyConvFilters * 8 * 8};
    const nvinfer1::Weights fcPolBiases{
        nvinfer1::DataType::kFLOAT, &weights.ip_pol_b[0], knumPolicyOutputs};
    auto fcPol = network->addFullyConnected(
        *reluPol->getOutput(0), knumPolicyOutputs, fcPolWeights, fcPolBiases);

    auto policyOut = network->addSoftMax(*fcPol->getOutput(0));
    nvinfer1::ITensor* policyOutTensor = policyOut->getOutput(0);
    policyOutTensor->setName("policyOut");

    // value head
    const int kvalueConvFilters = weights.value.bn_means.size();
    const nvinfer1::Weights convValWeights{
        nvinfer1::DataType::kFLOAT, &weights.value.weights[0],
        kvalueConvFilters * kNumFilters * 1 * 1};
    const nvinfer1::Weights convValBiases{nvinfer1::DataType::kFLOAT,
                                          &weights.value.biases[0],
                                          kvalueConvFilters};
    auto convVal = network->addConvolution(*resiOutTensor, kvalueConvFilters,
                                           nvinfer1::DimsHW{1, 1},
                                           convValWeights, convValBiases);
    auto reluVal = network->addActivation(*convVal->getOutput(0),
                                          nvinfer1::ActivationType::kRELU);

    const int kValueFcNodes = weights.ip1_val_b.size();
    const nvinfer1::Weights fcVal1Weights{
        nvinfer1::DataType::kFLOAT, &weights.ip1_val_w[0],
        kValueFcNodes * kvalueConvFilters * 8 * 8};
    const nvinfer1::Weights fcVal1Baises{nvinfer1::DataType::kFLOAT,
                                         &weights.ip1_val_b[0], kValueFcNodes};
    auto fcVal1 = network->addFullyConnected(
        *reluVal->getOutput(0), kValueFcNodes, fcVal1Weights, fcVal1Baises);
    auto reluValFc1 = network->addActivation(*fcVal1->getOutput(0),
                                             nvinfer1::ActivationType::kRELU);

    const nvinfer1::Weights fcVal2Weights{
        nvinfer1::DataType::kFLOAT, &weights.ip2_val_w[0], 1 * kValueFcNodes};
    const nvinfer1::Weights fcVal2Biases{nvinfer1::DataType::kFLOAT,
                                         &weights.ip2_val_b[0], 1};
    auto fcVal2 = network->addFullyConnected(*reluValFc1->getOutput(0), 1,
                                             fcVal2Weights, fcVal2Biases);

    auto valueOut = network->addActivation(*fcVal2->getOutput(0),
                                           nvinfer1::ActivationType::kTANH);
    nvinfer1::ITensor* valueOutTensor = valueOut->getOutput(0);
    valueOutTensor->setName("valueOut");

    network->markOutput(*valueOutTensor);
    network->markOutput(*policyOutTensor);

    // Ankan - test fp16
    // builder->setFp16Mode(true);

    builder->setInt8Mode(true);
    builder->setInt8Calibrator(&calibrator);

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    printf("\nGPU Memory, total: %lu, free: %lu\n", totalMem, freeMem);

    // reserve half of free mem for large batch, 1/4th for medium batch, and
    // 1/8th for small batch
    builder->setMaxBatchSize(kLargeBatchSize);
    builder->setMaxWorkspaceSize(freeMem / 2);
    builder->setMinFindIterations(2);
    builder->setAverageFindIterations(2);
    engine_large_ = builder->buildCudaEngine(*network);
    context_large_ = engine_large_->createExecutionContext();

    builder->setMaxBatchSize(kMediumBatchSize);
    builder->setMaxWorkspaceSize(freeMem / 4);
    engine_medium_ = builder->buildCudaEngine(*network);
    context_medium_ = engine_medium_->createExecutionContext();

    builder->setMaxBatchSize(kSmallBatchSize);
    builder->setMaxWorkspaceSize(freeMem / 8);
    engine_small_ = builder->buildCudaEngine(*network);
    context_small_ = engine_small_->createExecutionContext();

    // TODO: buildCudaEngine take a very long time (as TRT does all its
    // optimizations inside this function) Need to serialize engnie object and
    // save it to file (for each weight file), and load it directly from file if
    // present

    network->destroy();
    builder->destroy();
  }

  void forwardEval(InputsOutputs* io, int batchSize) {
    std::lock_guard<std::mutex> lock(lock_);

#ifdef DEBUG_RAW_NPS
    auto t_start = std::chrono::high_resolution_clock::now();
#endif

    // expand packed planes to full planes
    uint64_t* ipDataMasks = io->input_masks_mem_gpu_;
    float* ipDataValues = io->input_val_mem_gpu_;
    expandPlanes_Fp32_NCHW((float*)scratch_mem_, ipDataMasks, ipDataValues,
                           batchSize * kInputPlanes);

    float* opPol = io->op_policy_mem_gpu_;
    float* opVal = io->op_value_mem_gpu_;

    // Run the network using TRT.

    // decide which engine to use based on batch size
    nvinfer1::IExecutionContext* context;
    nvinfer1::ICudaEngine* engine;
    if (batchSize <= kSmallBatchSize) {
      context = context_small_;
      engine = engine_small_;
    } else if (batchSize <= kMediumBatchSize) {
      context = context_medium_;
      engine = engine_medium_;
    } else {
      context = context_large_;
      engine = engine_large_;
    }

    assert(engine->getNbBindings() == 3);
    void* buffers[3];

    const int inputIndex = engine->getBindingIndex("board");
    const int valueOutIndex = engine->getBindingIndex("valueOut");
    const int policyOutIndex = engine->getBindingIndex("policyOut");

    buffers[inputIndex] = scratch_mem_;
    buffers[valueOutIndex] = opVal;
    buffers[policyOutIndex] = opPol;

    context->enqueue(batchSize, buffers, (hipStream_t)0, nullptr);
    hipDeviceSynchronize();

#ifdef DEBUG_RAW_NPS
    const int reportingCalls = 100;
    static int numCalls = 0;
    static int sumBatchSize = 0;
    static double totalTime = 0;

    sumBatchSize += batchSize;
    numCalls++;

    auto t_end = std::chrono::high_resolution_clock::now();

    double dt = std::chrono::duration<double>(t_end - t_start).count();
    totalTime += dt;
    if (numCalls == reportingCalls) {
      double avgBatchSize = ((double)sumBatchSize) / numCalls;
      printf("\nAvg batch size: %lf, NN eval time: %lf seconds per %d evals\n",
             avgBatchSize, totalTime, sumBatchSize);
      sumBatchSize = 0;
      totalTime = 0;
      numCalls = 0;
    }
#endif
  }

  ~TRTNetwork() {
    if (scratch_mem_) ReportCUDAErrors(hipFree(scratch_mem_));

    context_small_->destroy();
    context_medium_->destroy();
    context_large_->destroy();

    engine_small_->destroy();
    engine_medium_->destroy();
    engine_large_->destroy();
  }

  std::unique_ptr<NetworkComputation> NewComputation() override {
    // set correct gpu id for this computation (as it might have been called
    // from a different thread)
    ReportCUDAErrors(hipSetDevice(gpu_id_));
    return std::make_unique<TRTNetworkComputation>(this);
  }

  std::unique_ptr<InputsOutputs> GetInputsOutputs() {
    std::lock_guard<std::mutex> lock(inputs_outputs_lock_);
    if (free_inputs_outputs_.empty()) {
      return std::make_unique<InputsOutputs>();
    } else {
      std::unique_ptr<InputsOutputs> resource =
          std::move(free_inputs_outputs_.front());
      free_inputs_outputs_.pop_front();
      return resource;
    }
  }

  void ReleaseInputsOutputs(std::unique_ptr<InputsOutputs> resource) {
    std::lock_guard<std::mutex> lock(inputs_outputs_lock_);
    free_inputs_outputs_.push_back(std::move(resource));
  }

 private:
  int gpu_id_;

  // TODO: maybe make these configurible
  static const int kSmallBatchSize = 16;
  static const int kMediumBatchSize = 64;
  static const int kLargeBatchSize = 1024;

  nvinfer1::IExecutionContext *context_small_, *context_medium_,
      *context_large_;
  nvinfer1::ICudaEngine *engine_small_, *engine_medium_, *engine_large_;

  // currently only one NN Eval can happen a time (we can fix this if needed by
  // allocating more memory)
  mutable std::mutex lock_;

  int numBlocks_;
  void* scratch_mem_;
  size_t scratch_size_;  // unused?

  mutable std::mutex inputs_outputs_lock_;
  std::list<std::unique_ptr<InputsOutputs>> free_inputs_outputs_;
};

TRTNetworkComputation::TRTNetworkComputation(TRTNetwork* network)
    : network_(network) {
  batch_size_ = 0;
  inputs_outputs_ = network_->GetInputsOutputs();
}

TRTNetworkComputation::~TRTNetworkComputation() {
  network_->ReleaseInputsOutputs(std::move(inputs_outputs_));
}

void TRTNetworkComputation::ComputeBlocking() {
  network_->forwardEval(inputs_outputs_.get(), GetBatchSize());
}

REGISTER_NETWORK("trt", TRTNetwork, 120)
}  // namespace lczero
